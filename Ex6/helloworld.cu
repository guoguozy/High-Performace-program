
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void helloworld(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    printf("Hello World from (%d,%d)!\n", i , j);
}
int main()
{
    dim3 grid(2,4);
    dim3 block(8,16);
    helloworld <<<grid,block>>>();
    hipDeviceSynchronize();
    return 0;
}
